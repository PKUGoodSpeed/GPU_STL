#include "hip/hip_runtime.h"
#include <iostream>
#include "../ginkgo/GOrderList.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>

using namespace std;

__global__ void test(){
    int pos = 0, ppos = 0, pnl = 0;
    // Creating an OrderList struct
    gpu_ginkgo::OrderList<100, 10> ggol(true, 1024, 10);
    ggol.getTime(1.5,  1.0);
    printf("<<< CREATING A NEW ORDER LIST STRUCTURE >>>\n");
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    int q_lim;
    // SENDING SELLING ORDERS
    printf("<<< SENDING NEW SELLING ORDERS >>>\n");
    q_lim = 2;
    ggol.sendNewSellingOrders(1029, 1033, q_lim);
    ggol.getTime(1.6, 1.0);
    q_lim = 2;
    ggol.sendNewSellingOrders(1029, 1033, q_lim);
    ggol.getTime(1.7, 1.0);
    q_lim = 2;
    ggol.sendNewSellingOrders(1030, 1034, q_lim);
    ggol.getTime(1.8, 1.0);
    q_lim = 2;
    ggol.sendNewSellingOrders(1031, 1035, q_lim);
    ggol.getTime(1.9, 1.0);
    q_lim = 37;
    ggol.sendNewSellingOrders(1029, 1033, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.updatePositions(pos);
    ggol.updatePnl(pnl);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // SENDING NEW BUYING ORDERS
    printf("<<< SENDING NEW BUYING ORDERS >>>\n");
    q_lim = 2;
    ggol.sendNewBuyingOrders(1028, 1032, q_lim);
    ggol.getTime(2.0, 1.0);
    q_lim = 3;
    ggol.sendNewBuyingOrders(1027, 1031, q_lim);
    ggol.getTime(2.1, 1.0);
    q_lim = 4;
    ggol.sendNewBuyingOrders(1026, 1030, q_lim);
    ggol.getTime(2.2, 1.0);
    q_lim = 17;
    ggol.sendNewBuyingOrders(1028, 1032, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.updatePositions(pos);
    ggol.updatePnl(pnl);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // 2.9s Update
    printf("<<< 2.9s BOOK UPDATES >>>\n");
    ggol.getTime(2.9, 0.17);
    int bz[10] = {10, 10, 15, 15, 20, 20, 15, 5, 25, 45};
    ggol.updatePendingOrders(bz);
    ggol.updateCancelOrders(1029, bz);
    ggol.updateAckedOrders(1029, bz);
    ggol.preCanceling(1030, 1032);
    q_lim = 37;
    ggol.sendNewSellingOrders(1029, 33, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.updatePositions(pos);
    ggol.updatePnl(pnl);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // 3.2s Update
    printf("<<< 3.2s BOOK UPDATES >>>\n");
    ggol.getTime(3.2, 0.17);
    for(int i=0;i<10;++i) bz[i] = (i+1)*10;
    ggol.updatePendingOrders(bz);
    ggol.updateCancelOrders(1029, bz);
    ggol.updateAckedOrders(1029, bz);
    ggol.preCanceling(1030, 1032);
    q_lim = 37;
    ggol.sendNewSellingOrders(1029, 1033, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.updatePositions(pos);
    ggol.updatePnl(pnl);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // A trade comes with price = 1030, qty = 90;
    printf("<<< TRADE: PRICE = 1030, QTY = 90 >>>\n");
    ggol.getTime(3.3, 0.17);
    int tv = 75;
    ggol.getTradedThrough(tv, 1030, bz);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.updatePositions(pos);
    ggol.updatePnl(pnl);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    for(int i=0;i<10;++i) printf("%d:  \t%d\n",1024+i, bz[i]);
    printf("\n");
    printf("--------------------------------------------------------\n\n");
    
    
    // Test finished
    printf("\n <<< TEST FINISHED !!! >>>\n");
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 1>>>();
    return 0;
}