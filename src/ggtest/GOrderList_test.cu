#include "hip/hip_runtime.h"
#include <iostream>
#include "../ginkgo/GOrderList.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>

using namespace std;

__global__ void test(){
    int pos = 0, ppos = 0, pnl = 0, tqty;
    // Creating an OrderList struct
    gpu_ginkgo::OrderList<100, 6> ggol(true, 1024, 10);
    ggol.getTime(1.5,  1.0);
    printf("<<< CREATING A NEW ORDER LIST STRUCTURE >>>\n");
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Sending new Orders
    printf("<<< SENDING FOUR NEW ORDERS >>>\n");
    int q_lim = 25;
    ggol.sendNewOrder(1024, q_lim);
    ggol.sendNewOrder(1024, q_lim);
    ggol.sendNewOrder(1025, q_lim);
    ggol.sendNewOrder(1026, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Sending new Orders
    printf("<<< SENDING ANOTHER ORDERS >>>\n");
    q_lim = 20;
    ggol.getTime(1.7, 1.0);
    ggol.sendNewOrder(1026, q_lim);
    ggol.sendNewOrder(1027, q_lim);
    ggol.sendNewOrder(1028, q_lim);
    ggol.sendNewOrder(1029, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Sending multiple orders to the last_level
    printf("<<< SENDING ANOTHER ORDERS >>>\n");
    ggol.getTime(2.0, 1.0);
    q_lim = 2;
    ggol.sendNewOrder(1029, q_lim);
    q_lim = 2;
    ggol.sendNewOrder(1029, q_lim);
    q_lim = 2;
    ggol.sendNewOrder(1029, q_lim);
    q_lim = 2;
    ggol.sendNewOrder(1029, q_lim);
    q_lim = 2;
    ggol.sendNewOrder(1029, q_lim);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Acking new orders
    ggol.getTime(2.6, 1.0);
    printf("<<< ACKING NEW ORDERS >>>");
    for(auto j=ggol.porders.begin(); j!=ggol.porders.end(); ){
        if(ggol.porders.at(j).acked_time < ggol.cur_time){
            ggol.ackPendingOrder(j, 10);
        }
        else ggol.porders.increment(j);
    }
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Canceling orders with price = 1026
    printf("<<< CANCELING ORDERS WITH PRICE = 1026 >>>");
    for(auto j=ggol.porders.begin(); j!=ggol.porders.end(); ){
        if(ggol.porders.at(j).price == 1026){
            ggol.cancelPendingOrder(j);
        }
        else ggol.porders.increment(j);
    }
    
    for(auto j=ggol.orders.begin(); j!=ggol.orders.end(); ){
        if(ggol.orders.at(j).price == 1026){
            ggol.cancelAckedOrder(j);
        }
        else ggol.orders.increment(j);
    }
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Swipe orders with price = 1024
    printf("<<< SWIPING ORDERS WITH PRICE = 1024 >>>");
    for(auto j=ggol.orders.begin(); j!=ggol.orders.end(); ){
        if(ggol.orders.at(j).price == 1024){
            ggol.swipeAckedOrder(j);
        }
        else ggol.orders.increment(j);
    }
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Acking new orders
    ggol.getTime(2.71, 1.0);
    printf("<<< ACKING NEW ORDERS >>>");
    for(auto j=ggol.porders.begin(); j!=ggol.porders.end(); ){
        if(ggol.porders.at(j).acked_time < ggol.cur_time){
            ggol.ackPendingOrder(j, 10);
        }
        else ggol.porders.increment(j);
    }
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // A trade comes with price = 1026, qty = 17
    printf("<<< A AGGRESSIVE TRADE COMES WITH PRICE = 1026, QTY = 17 >>>");
    int tv = 17, prc = 1026;
    int book_size[100];
    ggol.getTradedThrough(tv, prc, book_size);
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Canceling orders with price = 1029
    ggol.getTime(2.71, 0.1);
    printf("<<< CANCELING ORDERS WITH PRICE = 1029 >>>");
    for(auto j=ggol.porders.begin(); j!=ggol.porders.end(); ){
        if(ggol.porders.at(j).price == 1029){
            ggol.cancelPendingOrder(j);
        }
        else ggol.porders.increment(j);
    }
    
    for(auto j=ggol.orders.begin(); j!=ggol.orders.end(); ){
        if(ggol.orders.at(j).price == 1029){
            ggol.cancelAckedOrder(j);
        }
        else ggol.orders.increment(j);
    }
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Canceling orders with price = 1029
    ggol.getTime(3.91, 0.1);
    printf("<<< CLEAN UP CANCEL ORDERS >>>");
    for(auto j=ggol.corders.begin(); j!= ggol.corders.end(); ){
        if(ggol.corders.at(j).cancel_time < ggol.cur_time){
            ggol.cancelOrder(j);
        }
        else ggol.corders.increment(j);
    }
    ggol.showLevelQtyInfo();
    ggol.showPendingOrderInfo();
    ggol.showAckedOrderInfo();
    ggol.showCanceledOrderInfo();
    ggol.showUpdateInfo();
    ggol.reset(pos, pnl, tqty);
    printf("position = %d, pending position = %d, pnl = %d \n", pos, ppos, pnl);
    printf("--------------------------------------------------------\n\n");
    
    // Test finished
    printf("\n <<< TEST FINISHED !!! >>>\n");
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 1>>>();
    return 0;
}