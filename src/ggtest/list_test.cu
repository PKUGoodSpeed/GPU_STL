#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/lglist.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;

const int MAX_LENGTH = 170;

__device__ void printList(gpu_linearized_stl::list<float, MAX_LENGTH> &l){
    if(l.empty()){
        printf("\n===== Empty list =====\n\n");
        return;
    }
    printf("\n===== Showing the list =====\n");
    printf("list size = %d \n", l.size());
    for(auto i=l.begin(); i!=l.end(); l.increment(i)){
        printf("%.2f ", l.at(i));
    }
    printf("\n\n");
}


__global__ void test(float *output){
    gpu_linearized_stl::list<float, MAX_LENGTH> list;
    printList(list);
    //
    for(int i=0;i<5;++i) list.push_back(i*1.7);
    printf("for(int i=0;i<5;++i) list.push_back(i*1.7);");
    printList(list);
    //
    for(int i=6;i<10;++i) list.push_front(i*2.5);
    printf("for(int i=6;i<11;++i) list.push_front(i*2.5);");
    printList(list);
    //
    for(int i=0;i<2;++i) list.pop_back();
    printf("for(int i=0;i<2;++i) list.pop_back();");
    printList(list);
    
    //
    for(int i=0;i<2;++i) list.pop_front();
    printf("for(int i=0;i<2;++i) list.pop_front();");
    printList(list);
    //
    printf("list.at(list.begin())\n");
    printf("%.2f\n\n", list.at(list.begin()));
    //
    printf("list.front()\n");
    printf("%.2f\n\n", list.front());
    //
    auto idx = list.end();
    printf("auto idx = list.end(); list.at(list.decrement(idx))\n");
    printf("%.2f\n\n", list.at(list.decrement(idx)));
    //
    printf("list.back()\n");
    printf("%.2f\n\n", list.back());
    //
    list.reverse();
    printf("list.reverse();");
    printList(list);
    //
    list.insert(list.begin(), 3.14);
    printf("list.insert(list.begin(), 3.14);");
    printList(list);
    //
    list.insert(list.end(), 3.14);
    printf("list.insert(list.end(), 3.14);");
    printList(list);
    // 
    list.erase(list.begin());
    printf("list.erase(list.begin());");
    printList(list);
    //
    printf("list.at(list.begin())\n");
    printf("%.2f\n\n", list.at(list.begin()));
    //
    list.erase(list.end());
    printf("list.erase(list.end());");
    printList(list);
    //
    printf("list.back()\n");
    printf("%.2f\n\n", list.back());
    //
    for(auto p=list.begin(); p!=list.end(); list.increment(p)) list.insert(p, 22.22);
    printf("for(auto p=list.begin(); p!=list.end(); list.increment(p)) list.insert(p, 22.22);");
    printList(list);
    //
    int p;
    while((p=list.find(22.22))!=list.end()) list.erase(p);
    printf("while((auto p=list.find(22.22))!=list.end()) list.erase(p);");
    printList(list);
    
    gpu_linearized_stl::list<float, MAX_LENGTH> list2;
    list2.push_back(list.at(list.begin()));
    list.at(list.begin()) = 10086;
    printf("Reference checking");
    printList(list);
    printList(list2);
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 2>>>(to_ptr(dev_out));
    return 0;
}