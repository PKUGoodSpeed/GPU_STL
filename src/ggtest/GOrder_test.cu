#include "hip/hip_runtime.h"
#include <iostream>
#include "../ginkgo/GOrder.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>

using namespace std;


__global__ void test(){
    gpu_ginkgo::Order o(1024, 10, 180, 30);
    printf("\n\n===== Checking basic member functions =====\n\n");
    o.showOrderInfo();
    o.getAcked(150);
    printf("\nThe trade is acked!\n");
    o.showOrderInfo();
    o.qUpdateAgainstTrade(75, 15);
    o.showOrderInfo();
    o.qUpdateAgainstTrade(15, 15);
    o.showOrderInfo();
    int trade_size = 32;
    int dq = 0;
    int dqs = 0;
    int filled_qty = 0;
    int filled_pnl = 0;
    int bz = 0;
    o.filledAgainstTrade(trade_size, filled_qty, filled_pnl, dq, dqs, 1024, bz);
    o.showOrderInfo();
    printf("====Updating Info====\n");
    printf("dq = %d, dqs = %d, filled_qty = %d, filled_pnl = %d, trade_size = %d\n",
    dq, dqs, filled_qty, filled_pnl, trade_size);
    printf("\n\n===== Checking Q update =====\n\n");
    o.qUpdateAgainstBU(25);
    printf("Book Volume = 25\n");
    o.showOrderInfo();
    o.qUpdateAgainstBU(100);
    printf("\nBook Volume = 100\n");
    o.showOrderInfo();
    o.qUpdateAgainstBU(40);
    printf("\nBook Volume = 40\n");
    o.showOrderInfo();
    printf("\n\n===== Canceling this order =====\n\n");
    o.cancel(225. + 100);
    o.showOrderInfo();
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 1>>>();
    return 0;
}
