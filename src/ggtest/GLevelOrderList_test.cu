#include "hip/hip_runtime.h"
#include <iostream>
#include "../ginkgo/GLevelOrderList.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>

using namespace std;
typedef gpu_ginkgo::LevelOrderList<5> gglol;

__global__ void test(){
    gglol *p;
    p = new gglol(1024, true);
    printf("p = new gglol(1024);");
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    printf("\n\n<<< SENDING A NEW ORDER >>>");
    p->sendNewOrder(20, 3.0);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< SENDING A NEW ORDER >>>");
    p->sendNewOrder(20, 3.0);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< SENDING A NEW ORDER >>>");
    p->sendNewOrder(20, 4.0);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< SOME ORDER BEING ACKED >>>");
    p->procPendingOrders(20, 3.1);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< CANCELING THESE ORDERS >>>");
    p->preCancel(5.2);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< SOME ORDER BEING ACKED >>>");
    p->procPendingOrders(20, 4.1);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< SOME TRADE COMING >>>\n");
    int tv = 55, fqty = 0, fpnl = 0, dq = 0, dqs = 0, bz = 0;
    printf("Trade size = %d, filled qty = %d, filled pnl = %d", tv, fqty, fpnl);
    p->procTrade(tv, fqty, fpnl, dq, dqs, 1024, bz);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    printf("Trade size = %d, filled qty = %d, filled pnl = %d\n\n", tv, fqty, fpnl);
    
    printf("\n\n<<< BOOK UPDATE OCCURS >>>");
    p->adjustQAgainstBU(100);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< BOOK UPDATE OCCURS >>>");
    p->adjustQAgainstBU(20);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    printf("\n\n<<< CANCEL TIME COMES >>>");
    p->procCanceledOrders(5.5);
    p->showLevelInfo();
    p->showPendingOrderInfo();
    p->showAckedOrderInfo();
    //
    
    delete p;
    p = NULL;
    if(!p) printf("\n\n SUCCESSFUL REMOVE THE LEVEL\n\n");
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 1>>>();
    return 0;
}