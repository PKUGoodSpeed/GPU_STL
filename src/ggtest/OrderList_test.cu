#include "hip/hip_runtime.h"
#include <iostream>
#include "../ginkgo/GOrder.h"
#include "../include/lglist.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>

using namespace std;
const int MAX_LENGTH = 20;

typedef gpu_ginkgo::Order gorder;

__device__ void printList(gpu_linearized_stl::list<gpu_ginkgo::Order, MAX_LENGTH> &ol){
    if(ol.empty()){
        printf("\n===== This is an empty order list =====\n\n");
        return;
    }
    for(auto p=ol.begin(); p!=ol.end(); ol.increment(p)) ol.at(p).showOrderInfo();
    printf("\n");
}

__global__ void test(){
    gpu_linearized_stl::list<gpu_ginkgo::Order, MAX_LENGTH> ol;
    printList(ol);
    //
    printf("Adding New Order\n");
    ol.push_back(gorder(1024, 15, 100, 0));
    printf("ol.push_back(gorder(1024, 15, 100, 0));\n");
    printList(ol);
    //
    printf("Adding New Order\n");
    ol.push_back(gorder(1024, 15, 101, 15));
    printf("ol.push_back(gorder(1024, 15, 101, 15));\n");
    printList(ol);
    //
    printf("Adding New Order\n");
    ol.push_back(gorder(1024, 15, 102, 30));
    printf("ol.push_back(gorder(1024, 15, 102, 30));\n");
    printList(ol);
    //
    printf("Adding New Order\n");
    ol.push_back(gorder(1024, 15, 103, 45));
    printf("ol.push_back(gorder(1024, 15, 103, 45));\n");
    printList(ol);
    //
    int p = ol.begin();
    printf("Acking The first Order\n");
    ol.at(p).getAcked(25);
    printf("int p = ol.begin();ol.at(p).getAcked(25);");
    printList(ol);
    //
    ol.at(p).qUpdateAgainstBU(50);
    ol.increment(p);
    ol.at(p).getAcked(50);
    printf("ol.at(p).qUpdateAgainstBU(50);ol.increment(p);ol.at(p).getAcked(50);");
    printList(ol);
    //
    ol.at(ol.begin()).qUpdateAgainstBU(75);
    ol.at(p).qUpdateAgainstBU(75);
    ol.increment(p);
    ol.at(p).getAcked(75);
    printf("ol.at(p).qUpdateAgainstBU(75);ol.increment(p);ol.at(p).getAcked(75);");
    printList(ol);
    //
    int j = ol.begin();
    for(int i=0;i<3;i++,ol.increment(j)) ol.at(j).qUpdateAgainstBU(100);
    ol.at(j).getAcked(100);
    printf("for(int i=0, j= ol.begin();i<3;i++,ol.increment(j)) ol.at(j).qUpdateAgainstBU(100);ol.at(j).getAcked(100);");
    printList(ol);
    //
    printf("Book Volume increases to 150\n");
    j = ol.begin();
    for(int i=0;i<4;i++,ol.increment(j)) ol.at(j).qUpdateAgainstBU(150);
    printf("for(int i=0;i<4;i++,ol.increment(j)) ol.at(j).qUpdateAgainstBU(150);");
    printList(ol);
    //
    printf("Book Volume decreases to 45\n");
    j = ol.begin();
    for(int i=0;i<4;i++,ol.increment(j)) ol.at(j).qUpdateAgainstBU(45);
    printf("for(int i=0;i<4;i++,ol.increment(j)) ol.at(j).qUpdateAgainstBU(45);");
    printList(ol);
    //
    printf("A trade of 60 with price == 1024 is coming against us!!!\n");
    int filled_qty = 0, filled_pnl = 0, dq = 0, dqs = 0, tz = 60, prc = 1024;
    int bz = 0;
    
    for(j=ol.begin(); j!=ol.end();){
        if(ol.at(j).price != prc){
            ol.increment(j);
            continue;
        }
        ol.at(j).qUpdateAgainstTrade(dq, dqs);
        if(ol.at(j).filledAgainstTrade(tz, filled_qty, filled_pnl, dq, dqs, 1024, bz)){
            j = ol.erase(j);
        }
        else ol.increment(j);
    }
    printList(ol);
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 1>>>();
    return 0;
}