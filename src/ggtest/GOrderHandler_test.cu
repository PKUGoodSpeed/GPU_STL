#include "hip/hip_runtime.h"
#include <iostream>
#include "../ginkgo/GOrderHandler.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>

using namespace std;

__global__ void test(){
    // Creating an OrderHandler struct
    gpu_ginkgo::OrderHandler<100, 10> ggoh(1024, 10);
    ggoh.showOrderBookInfo();
    ggoh.loadStrategy(33, 0., 0.);
    
    // Start Updating Book
    printf("{{{{{{{{{{{{{  NEW BOOK UPDATE }}}}}}}}}}}}}\n");
    int bz1[10] = {1,2,3,4,5,5,4,3,2,1};
    int ap1 = 1029, bp1 = 1028;
    double mp1 = 1028.5;
    ggoh.getTimeInfo(1., 0.2);
    ggoh.bookUpdateSim(bz1, ap1, bp1, mp1);
    ggoh.cancelAndSendNewOrders();
    ggoh.showBasicInfo();
    ggoh.showOrderBookInfo();
    
    // New book Update
    printf("{{{{{{{{{{{{{  NEW BOOK UPDATE }}}}}}}}}}}}}\n");
    int bz2[10] = {1, 3, 5, 7, 9, 11, 11, 9, 7, 5};
    int ap2 = 1030, bp2 = 1029;
    double mp2 = 1029.5;
    ggoh.getTimeInfo(1.3, 0.2);
    ggoh.bookUpdateSim(bz2, ap2, bp2, mp2);
    ggoh.cancelAndSendNewOrders();
    ggoh.showBasicInfo();
    ggoh.showOrderBookInfo();
    
    // New book Update
    printf("{{{{{{{{{{{{{  NEW BOOK UPDATE }}}}}}}}}}}}}\n");
    int bz3[10] = {3, 23, 4, 2, 3, 1, 9, 9, 7, 5};
    int ap3 = 1032, bp3 = 1031;
    double mp3 = 1031.5;
    ggoh.getTimeInfo(1.3, 0.2);
    ggoh.bookUpdateSim(bz3, ap3, bp3, mp3);
    ggoh.cancelAndSendNewOrders();
    ggoh.showBasicInfo();
    ggoh.showOrderBookInfo();
    
    // trade
    printf("{{{{{{{{{{{{{  TRADE COMES }}}}}}}}}}}}}\n");
    int tv = 32;
    ggoh.getTimeInfo(1.6, 0.2);
    ggoh.processTrade(true, 1028, tv);
    ggoh.cancelAndSendNewOrders();
    ggoh.showBasicInfo();
    ggoh.showOrderBookInfo();
    
    // Test finished
    printf("\n <<< TEST FINISHED !!! >>>\n");
}

int main(){
    def_dvec(float) dev_out(1, 0);
    test<<<1, 1>>>();
    return 0;
}