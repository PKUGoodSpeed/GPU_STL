#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/glist.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;


__global__ void test(float *output){
    gpu_stl::list<float> list;
    int idx = 0;
    output[idx++] = (float)list.empty();
    output[idx++] = (float)list.size();
    for(int i=0;i<10;++i){
        list.push_back(i*1.7);
        output[idx++] = (float)list.empty();
        output[idx++] = (float)list.size();
    }
    for(int i=0;i<6;++i) {
        auto p = list.insert(--list.end(), 55);
        ++p;
        ++p;
        list.insert(p, 77);
    }
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    output[idx++] = list.front();
    output[idx++] = *list.begin();
    *list.begin() = 3.1415926;
    output[idx++] = list.front();
    output[idx++] = *list.begin();
    output[idx++] = 10086;
    output[idx++] = list.back();
    output[idx++] = *(--list.end());
    *(--list.end()) = 3.124235;
    output[idx++] = list.back();
    output[idx++] = *(--list.end());
    gpu_stl::list<float>::iterator p;
    while((p=list.find(77))!=list.end()){
        list.erase(p);
    }
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    list.reverse();
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    list.pop_front();
    list.pop_back();
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    gpu_stl::list<float>::iterator p1(--list.end());
    output[idx++] = 10086;
    output[idx++] = *p1;
    list.clear();
    output[idx++] = 10086;
    output[idx++] = list.empty();
    output[idx++] = 10086;
    output[idx++] = list.size();
    output[idx++] = max(123141, 2335436);
    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);
}

int main(){
    def_dvec(float) dev_out(120, 0);
    test<<<10, 10>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
