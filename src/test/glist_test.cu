#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/glist.hpp"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;


__global__ void test(float *output){
    gpu_stl::list<float> list;
    int idx = 0;
    output[idx++] = (float)list.empty();
    output[idx++] = (float)list.size();
    for(int i=0;i<10;++i){
        list.push_back(i*1.7);
        output[idx++] = (float)list.empty();
        output[idx++] = (float)list.size();
    }
    for(int i=0;i<6;++i) {
        auto p = list.insert(--list.end(), 55);
        ++p;
        ++p;
        list.insert(p, 77);
    }
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    output[idx++] = list.front();
    output[idx++] = *list.begin();
    *list.begin() = 3.1415926;
    output[idx++] = list.front();
    output[idx++] = *list.begin();
    output[idx++] = 10086;
    output[idx++] = list.back();
    output[idx++] = *(--list.end());
    *(--list.end()) = 3.124235;
    output[idx++] = list.back();
    output[idx++] = *(--list.end());
    gpu_stl::list<float>::iterator p;
    while((p=list.find(77))!=list.end()){
        list.erase(p);
    }
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    list.reverse();
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
    list.pop_front();
    list.pop_back();
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();++p) output[idx++] = *p;
}

int main(){
    def_dvec(float) dev_out(120, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
