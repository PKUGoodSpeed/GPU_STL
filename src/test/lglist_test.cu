#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/lglist.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;


__global__ void test(float *output){
    gpu_linearized_stl::list<float,100> list;
    int idx = 0;
    output[idx++] = list.max_size();
    output[idx++] = list.full();
    output[idx++] = (float)list.empty();
    output[idx++] = (float)list.size();
    for(int i=0;i<10;++i){
        list.push_back(i*1.7);
        output[idx++] = (float)list.empty();
        output[idx++] = (float)list.size();
    }
    for(int i=0;i<6;++i) {
        int a = list.end();
        auto p = list.insert(list.decrement(a), 55);
        list.decrement(p);
        list.decrement(p);
        list.insert(p, 77);
    }
    for(auto p=list.begin(); p!=list.end();list.increment(p)) output[idx++] = list.at(p);
    output[idx++] = list.front();
    output[idx++] = list.back();
    int p;
    while((p=list.find(77))!=list.end()){
        list.erase(p);
    }
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();list.increment(p)) output[idx++] = list.at(p);
    list.reverse();
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();list.increment(p)) output[idx++] = list.at(p);
    list.pop_front();
    list.pop_back();
    list.pop_back();
    output[idx++] = 10086;
    for(auto p=list.begin(); p!=list.end();list.increment(p)) output[idx++] = list.at(p);
}

int main(){
    def_dvec(float) dev_out(150, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
