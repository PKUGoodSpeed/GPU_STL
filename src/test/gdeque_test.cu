#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/gdeque.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;


__global__ void test(float *output){
    gpu_stl::deque<float> deque;
    int idx = 0;
    output[idx++] = deque.empty();
    output[idx++] = deque.size();
    output[idx++] = 10086;
    for(int i=1;i<=15;++i){
        if(i%2) deque.push_front(i*1.7);
        else deque.push_back(i*1.7);
        output[idx++] = deque.empty();
        output[idx++] = deque.size();
    }
    output[idx++] = 10086;
    while(!deque.empty()){
        output[idx++] = deque.empty();
        output[idx++] = deque.size();
        output[idx++] = deque.front();
        output[idx++] = deque.back();
        deque.pop_front();
    }
}

int main(){
    def_dvec(float) dev_out(120, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
