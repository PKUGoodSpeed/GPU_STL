#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/lgstack.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;


__global__ void test(float *output){
    gpu_linearized_stl::stack<float, 40> stk;
    int idx = 0;
    output[idx++] = stk.max_size();
    output[idx++] = stk.empty();
    output[idx++] = stk.size();
    output[idx++] = 10086;
    for(int i=1;i<=20;++i){
        stk.push(i*1.7);
        output[idx++] = stk.empty();
        output[idx++] = stk.size();
    }
    output[idx++] = 10086;
    while(!stk.empty()){
        output[idx++] = stk.empty();
        output[idx++] = stk.size();
        output[idx++] = stk.top();
        stk.pop();
    }
}

int main(){
    def_dvec(float) dev_out(120, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
