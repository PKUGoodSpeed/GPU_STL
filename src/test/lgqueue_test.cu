#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/lgqueue.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;


__global__ void test(float *output){
    gpu_linearized_stl::queue<float,75> que;
    int idx = 0;
    output[idx++] = que.max_size();
    output[idx++] = que.empty();
    output[idx++] = que.size();
    output[idx++] = 10086;
    for(int i=1;i<=15;++i){
        que.push(i*1.7);
        output[idx++] = que.empty();
        output[idx++] = que.size();
    }
    output[idx++] = 10086;
    while(!que.empty()){
        output[idx++] = que.empty();
        output[idx++] = que.size();
        output[idx++] = que.front();
        output[idx++] = que.back();
        que.pop();
    }
}

int main(){
    def_dvec(float) dev_out(120, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
