#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/gpu_queue.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;
const int MAX_QUEUE_SIZE = 50;

__global__ void test(float *output){
    gpu_queue<float, MAX_QUEUE_SIZE> que;
    for(int i=1;i<=MAX_QUEUE_SIZE;++i){
        que.push(1.7*i);
    }
    int idx = 0, k = 0;
    while(!que.empty()){
        que.pop_k(k);
        if(que.empty()) return;
        output[idx] = que.front();
        idx += 1;
        output[idx] = que.back();
        k += 1;
        idx += 1;
    }
}

int main(){
    def_dvec(float) dev_out(40, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}
