#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/gpu_map.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;

const int MAP_SIZE = 100;

__global__ void test(float *output){
    gpu_map<int, float, MAP_SIZE> map;
    for(int i=0;i<MAP_SIZE;++i){
        map.set(2*i, 1.7*i);
    }
    int idx = 0;
    for(int i=0;i<100;++i){
        output[idx++] = map.get(i);
        output[idx++] = (int)map.rget(1.7*i);
        map.erase(i);
        if(i%2) map.set(i, i*1.7);
    }
    for(int i=0;i<100;++i) {
        output[idx++] = map.get(i);
        output[idx++] = (int)map.rget(1.7*i);
    }
}

int main(){
    def_dvec(float) dev_out(400, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}