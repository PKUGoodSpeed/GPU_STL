#include "hip/hip_runtime.h"
#include <iostream>
#include <set>
#include "../include/gpu_set.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;

const int SET_SIZE = 100;

__global__ void test(int *output){
    gpu_set<int, SET_SIZE> set;
    for(int i=0;i<SET_SIZE;++i){
        set.insert(2*i);
    }
    int idx = 0;
    for(int i=0;i<100;++i){
        output[idx++] = i;
        output[idx++] = set.find(i);
        set.erase(i);
        if(i%2) set.insert(i);
    }
    for(int i=0;i<100;++i) {
        output[idx++] = i;
        output[idx++] = set.find(i);
    }
}

int main(){
    def_dvec(int) dev_out(400, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}