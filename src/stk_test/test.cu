#include "hip/hip_runtime.h"
#include <iostream>
#include "../include/gpu_stack.h"
#include <thrust/device_vector.h>
#define def_dvec(t) thrust::device_vector<t>
#define to_ptr(x) thrust::raw_pointer_cast(&x[0])
using namespace std;

const int STACK_SIZE = 100;

__global__ void test(float *output){
    gpu_stack<float, STACK_SIZE> stk;
    for(int i=1;i<=STACK_SIZE;++i){
        stk.push(1.5*i);
    }
    int idx = 0, k = 0;
    while(!stk.empty()){
        stk.pop_k(k);
        if(stk.empty()) return;
        output[idx] = stk.top();
        idx += 1;
        output[idx] = (float)stk.size();
        k += 1;
        idx += 1;
    }
}

int main(){
    def_dvec(float) dev_out(40, 0);
    test<<<1, 1>>>(to_ptr(dev_out));
    for(auto k:dev_out) cout<<k<<' ';
    cout<<endl;
    return 0;
}